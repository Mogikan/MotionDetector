#include "hip/hip_runtime.h"

#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>



//        var imagePixel = (byte)(0.3 * imagePixels[i * stride + j * _bytesPerPixel] + 0.59 * imagePixels[i][j * _bytesPerPixel + 1] + 0.11 * imagePixels[i][j * _bytesPerPixel + 2]);
//        _median[i][j] = (byte)(_median[i][j] + Math.Sign(imagePixel - _median[i][j]));//consult atricle
//        _delta[i][j] = (byte)(Math.Abs(imagePixel - _median[i][j]));
//        if (_delta[i][j] > 0.001)
//        {
//            _variance[i][j] = _variance[i][j] + Math.Sign(N * _delta[i][j] - _variance[i][j]);
//        }
//        if (_delta[i][j] > 0.001)
//        {
//            _detectionLabel[i][j] = (byte)(Convert.ToByte(_delta[i][j] >= _variance[i][j]) * 255);
//        }
//        else
//        {
//            _detectionLabel[i][j] = 0;
//        }


const int N = 4;
extern "C" __global__ void motionKernel(const unsigned char *imagePixels, unsigned char *median, unsigned char* delta, int* variance, unsigned char* motionBytes, unsigned int bytesPerPixel)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int imagePixel = (int)(0.3f * imagePixels[index * bytesPerPixel] + 0.59f * imagePixels[index * bytesPerPixel + 1] + 0.11f * imagePixels[index * bytesPerPixel + 2]);
	int medianDiff = (imagePixel - median[index]);
	median[index] +=  (medianDiff>0)?1:((medianDiff<0)?-1:0);
	int absdelta = abs(imagePixel - median[index]);
	delta[index] = (unsigned char)absdelta;
	if (absdelta > 0)
	{		
		int varianceChange = (N * absdelta - variance[index]);
		variance[index] = variance[index] + (varianceChange>0)?1:((varianceChange<0)?-1:0);
		motionBytes[index] = 255 * (absdelta >= variance[index]);
	}
	else
	{
		motionBytes[index] = 0;
	}
}


	//_median[i][j] = (byte)(_median[i][j] + Math.Sign(imagePixel - _median[i][j]));//consult atricle
	//_delta[i][j] = (byte)(Math.Abs(imagePixel - _median[i][j]));
	//
	//if (_delta[i][j] > 0.001)
	//{
	//	_variance[i][j] = _variance[i][j] + Math.Sign(N * _delta[i][j] - _variance[i][j]);
	//	_detectionLabel[i][j] = (byte)(Convert.ToByte(_delta[i][j] >= _variance[i][j]) * 255);
	//}
	//else
	//{
	//	_detectionLabel[i][j] = 0;
	//}


//int main()
//{
//	const int arraySize = 5;
//	const int a[arraySize] = { 1, 2, 3, 4, 5 };
//	const int b[arraySize] = { 10, 20, 30, 40, 50 };
//	int c[arraySize] = { 0 };
//
//	// Add vectors in parallel.
//	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addWithCuda failed!");
//		return 1;
//	}
//
//	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//		c[0], c[1], c[2], c[3], c[4]);
//
//	// hipDeviceReset must be called before exiting in order for profiling and
//	// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	cudaStatus = hipDeviceReset();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceReset failed!");
//		return 1;
//	}
//	fgetc(stdin);
//	return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//	//var imagePixel = (byte)(0.3 * imagePixels[i][j* _bitsPerPixel] + 0.59 * imagePixels[i][j * _bitsPerPixel + 1] + 0.11 * imagePixels[i][j * _bitsPerPixel + 2]);
//	int *dev_a = 0;
//	int *dev_b = 0;
//	int *dev_c = 0;
//	hipError_t cudaStatus;
//
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = hipSetDevice(0);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		goto Error;
//	}
//
//	// Allocate GPU buffers for three vectors (two input, one output)    .
//	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//	
//
//	// Check for any errors launching the kernel
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// hipDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		goto Error;
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//Error:
//	hipFree(dev_c);
//	hipFree(dev_a);
//	hipFree(dev_b);	
//	return cudaStatus;
//}
